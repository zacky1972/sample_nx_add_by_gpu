#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "vectorAdd.h"

__global__ void vectorAdd(const int32_t *A, const int32_t *B, int32_t *C, uint64_t numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

#ifdef __cplusplus
extern "C" {
#endif
bool add_s32_cuda(const int32_t *h_A, const int32_t *h_B, int32_t *h_C, uint64_t numElements, char *error_message)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    
    // compute numElements
    uint64_t size = numElements * sizeof(int32_t);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        snprintf(error_message, MAXBUFLEN, "h_A, h_B and h_C must be not NULL.");
        return false;
    }

    // Allocate the device input vector A
    int32_t *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        snprintf(error_message, MAXBUFLEN, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
        return false;
    }

    // Allocate the device input vector B
    int32_t *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess) {
        snprintf(error_message, MAXBUFLEN, "Failed to allocate device vector B (error code %s)!\n",
            hipGetErrorString(err));
        return false;
    }


    // Allocate the device input vector C
    int32_t *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess) {
        snprintf(error_message, MAXBUFLEN, "Failed to allocate device vector C (error code %s)!\n",
            hipGetErrorString(err));
        return false;
    }

    // Copy the host input vectors A and B in host memory to the device input
    // vectors in
    // device memory
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        snprintf(error_message, MAXBUFLEN,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
        return false;
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        snprintf(error_message, MAXBUFLEN,
            "Failed to copy vector B from host to device (error code %s)!\n",
            hipGetErrorString(err));
        return false;
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
    //     threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess) {
        snprintf(error_message, MAXBUFLEN, "Failed to launch vectorAdd kernel (error code %s)!\n",
             hipGetErrorString(err));
        return false;
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    // printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        snprintf(error_message, MAXBUFLEN,
             "Failed to copy vector C from device to host (error code %s)!\n",
             hipGetErrorString(err));
        return false;
    }

    // Free device global memory
    err = hipFree(d_A);
    if (err != hipSuccess) {
        snprintf(error_message, MAXBUFLEN, "Failed to free device vector A (error code %s)!\n",
             hipGetErrorString(err));
        return false;
    }
    err = hipFree(d_B);
    if (err != hipSuccess) {
        snprintf(error_message, MAXBUFLEN, "Failed to free device vector B (error code %s)!\n",
             hipGetErrorString(err));
        return false;
    }
    err = hipFree(d_C);
    if (err != hipSuccess) {
        snprintf(error_message, MAXBUFLEN, "Failed to free device vector C (error code %s)!\n",
             hipGetErrorString(err));
        return false;
    }

    return true;
}
#ifdef __cplusplus
}
#endif
